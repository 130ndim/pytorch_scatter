#include "hip/hip_runtime.h"
#include <THC/THC.h>

#include "kernel.h"
#include "common.cuh"

#define scatter_(NAME) TH_CONCAT_4(scatter_, NAME, _kernel_, Real)
#define index_backward TH_CONCAT_2(index_backward_kernel_, Real)
#define check TH_CONCAT_2(check_kernel_, Real)

#define thc_(NAME) TH_CONCAT_4(thc_, NAME, _, Real)

#include "generic/common.cu"
#include "THCGenerateAllTypes.h"

template <typename Real, int Dims>
__global__ void maxKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, TensorInfo<int64_t> arg_output, const int dim, const int n) {
  KERNEL_LOOP(i, n) {

  }
}

#include "generic/kernel.cu"
#include "THCGenerateAllTypes.h"

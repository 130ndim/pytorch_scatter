#include "hip/hip_runtime.h"
#include <THC/THC.h>

#include "kernel.h"

#include "common.cuh"
#include "THCIndex.cuh"
#include "THCAtomics.cuh"

#define scatter_(NAME) TH_CONCAT_4(scatter_, NAME, _kernel_, Real)
#define index_backward TH_CONCAT_2(index_backward_kernel_, Real)
#define thc_(NAME) TH_CONCAT_4(thc_, NAME, _, Real)

#include "generic/common.cu"
#include "THCGenerateAllTypes.h"

template<typename Real, int Dims>
__global__ void maxKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, TensorInfo<int64_t> arg, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int inputOffset = 0;
    IndexToScatterOffsets<Real, Real, Dims>::compute(i, dim, index, &indexOffset, input, &inputOffset, output, &outputOffset);
    atomicMax(&output.data[outputOffset], input.data[inputOffset]);
    // TODO: Do something with arg.
  }
}

#include "generic/kernel.cu"
#include "THCGenerateAllTypes.h"

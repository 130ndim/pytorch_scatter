#include "hip/hip_runtime.h"
#include <THC/THC.h>

#include "kernel.h"

#include "common.cuh"
#include "THCIndex.cuh"
#include "THCAtomics.cuh"

#define scatter_(NAME) TH_CONCAT_4(scatter_, NAME, _kernel_, Real)
#define index_backward TH_CONCAT_2(index_backward_kernel_, Real)
#define thc_(NAME) TH_CONCAT_4(thc_, NAME, _, Real)

#include "generic/common.cu"
#include "THCGenerateAllTypes.h"

template<typename Real, int Dims>
__global__ void maxKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int inputOffset = 0;;
    IndexToScatterOffsets3<Real, Real, Dims>::compute(i, dim, index, &indexOffset, input, &inputOffset, output, &outputOffset);
    atomicMax(&output.data[outputOffset], input.data[inputOffset]);
  }
}

template<typename Real, int Dims>
__global__ void minKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int inputOffset = 0;;
    IndexToScatterOffsets3<Real, Real, Dims>::compute(i, dim, index, &indexOffset, input, &inputOffset, output, &outputOffset);
    atomicMin(&output.data[outputOffset], input.data[inputOffset]);
  }
}

template<typename Real, int Dims>
__global__ void argKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, TensorInfo<int64_t> arg, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int inputOffset = 0; int argOffset = 0;
    IndexToScatterOffsets4<Real, Real, int64_t, Dims>::compute(i, dim, index, &indexOffset, input, &inputOffset, output, &outputOffset, arg, &argOffset);
    if (eq(input.data[inputOffset], output.data[outputOffset])) arg.data[argOffset] = inputOffset % input.size[dim];
  }
}

#include "generic/kernel.cu"
#include "THCGenerateAllTypes.h"

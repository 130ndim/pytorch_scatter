
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

void scatter_(mul)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  thc_(check)(state, output, index, input);
  printf("mul");
}

void scatter_(div)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  thc_(check)(state, output, index, input);
  printf("div");
}

void scatter_(mean)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCTensor *num_output) {
  thc_(check)(state, output, index, input);
  printf("mean");
}

void scatter_(max)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCudaLongTensor *arg_output) {
  thc_(check)(state, output, index, input);

  const int n = THCudaLongTensor_nElement(state, index);
  TensorInfo<real> outputInfo = thc_(getTensorInfo)(state, output);
  TensorInfo<int64_t> indexInfo = thc_getTensorInfo_Long(state, index);
  TensorInfo<real> inputInfo = thc_(getTensorInfo)(state, input);
  TensorInfo<int64_t> argInfo = thc_getTensorInfo_Long(state, arg_output);

  KERNEL_RUN(maxKernel, indexInfo.dims, n, outputInfo, indexInfo, inputInfo, argInfo, dim)
  /* KERNEL_RUN(argKernel, indexInfo.dims, n, outputInfo, indexInfo, dim) */

  /* maxKernel<real, -1><<<GET_BLOCKS(n), NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(outputInfo, indexInfo, inputInfo, dim, n); */
  /* argKernel<real, -1><<<GET_BLOCKS(n), NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(dim, n); */
}

void scatter_(min)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCudaLongTensor *arg_output) {
  thc_(check)(state, output, index, input);
  printf("min");
}

void index_backward(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *grad, THCudaLongTensor *arg_grad) {
  thc_(check)(state, output, index, grad);
  printf("index_backward");
}

#endif

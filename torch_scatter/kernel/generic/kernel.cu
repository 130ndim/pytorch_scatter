
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

void scatter_(mul)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  printf("mul");
}

void scatter_(div)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  printf("div");
}

void scatter_(mean)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCTensor *num_output) {
  printf("mean");
}

void scatter_(max)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCudaLongTensor *arg_output) {
  printf("max");
}

void scatter_(min)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCudaLongTensor *arg_output) {
  printf("min");
}

void index_backward(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *grad, THCudaLongTensor *arg_grad) {
  printf("index_backward");
}

#endif

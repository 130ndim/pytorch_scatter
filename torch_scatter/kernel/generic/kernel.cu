
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

void scatter_(mul)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, output, input));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 2, index));
  THArgCheck(THCTensor_(nDimension)(state, output) <= MAX_DIMS, 1, "Tensor too large or too many dimensions");

  printf("mul");
}

void scatter_(div)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  printf("div");
}

void scatter_(mean)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCTensor *num_output) {
  printf("mean");
}

void scatter_(max)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCudaLongTensor *arg_output) {
  printf("max");
}

void scatter_(min)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCudaLongTensor *arg_output) {
  printf("min");
}

void index_backward(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *grad, THCudaLongTensor *arg_grad) {
  printf("index_backward");
}

#endif
